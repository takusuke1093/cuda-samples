#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 3600 
#define PI 3.14159265358979323846
#define DEG_TO_RAD(deg)  ((deg) / 180.0 * (PI))

__global__ void matrix_vector_multi_gpu_1_1(double *A_d)
{
	double deg = 0.0;	
	for (int i=0; i<=N; i+=1) {
		double a = __cosf(DEG_TO_RAD(deg));
		double b = cos(DEG_TO_RAD(deg));
		if (a != b) {
			A_d[i] = a - b;
			printf("%.16f %.16f \n", a,b );
		} else {
			A_d[i] = 100;
		}
		deg+=0.1;
	}
}


int main()
{
	int i;
	double A[N];   //HOST
	double *A_d;     //DEVICE
	FILE *outputfile;

	outputfile = fopen("output_sfu_cosine.txt", "w"); 
	if (outputfile == NULL) {
		printf("cannot open file! \n");
		exit(1);
	}

        dim3 blocks(1,1,1);
        dim3 threads(1,1,1);

	hipMalloc( (void**) &A_d, N*sizeof(double));
	
	hipMemcpy(A_d, A, N*sizeof(double), hipMemcpyHostToDevice); 
	
	matrix_vector_multi_gpu_1_1<<< blocks, threads >>>(A_d);

        hipMemcpy(A, A_d, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0;i<=N;i+=1){
		if (A[i] <  1 && A[i] > -1) fprintf(outputfile,"%d %.16f \n", i, A[i]);
	}

	fclose(outputfile);

        hipFree(A_d);

    return 0;
}
