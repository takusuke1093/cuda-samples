#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 3600 
#define PI 3.14159265358979323846
#define DEG_TO_RAD(deg)  ((deg) / 180.0 * (PI))

__global__ void sfu_sine_function(double *B_d, double *radius_d)
{
	for (int i = 0; i<=N; i+=1) {
		B_d[i] = __sinf(radius_d[i]);
	}
}


int main()
{
	int i;
	double B[N];   // HOST
	double radius[N];  // HOST
	double *B_d;   // DEVICE
	double *radius_d;  // DEVICE
	double deg = 0.0;
	FILE *outputfile;

	outputfile = fopen("output_sine.txt", "w"); 
	if (outputfile == NULL) {
		printf("cannot open file! \n");
		exit(1);
	}

	for (i = 0; i <= N; i++) {
		radius[i] = DEG_TO_RAD(deg);
		deg += 0.1;
	}

        dim3 blocks(1,1,1);
        dim3 threads(1,1,1);

	hipMalloc( (void**) &B_d, N*sizeof(double));
	hipMalloc( (void**) &radius_d, N*sizeof(double));
	
	hipMemcpy(B_d, B, N*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy(radius_d, radius, N*sizeof(double), hipMemcpyHostToDevice); 
	
	sfu_sine_function<<< blocks, threads >>>(B_d, radius_d);

        hipMemcpy(B, B_d, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0;i<=N;i+=1){
		fprintf(outputfile,"%d %.16f\n",i, B[i]);
	}

	fclose(outputfile);

        hipFree(B_d);
        hipFree(radius_d);

    return 0;
}

