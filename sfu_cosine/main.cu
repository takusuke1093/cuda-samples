#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 3600 
#define PI 3.14159265358979323846
#define DEG_TO_RAD(deg)  ((deg) / 180.0 * (PI))

__global__ void normal_cosine_function(double *B_d, double *radius_d)
{
	for (int i = 0; i<=N; i+=1) {
		B_d[i] = __cosf(radius_d[i]);
	}
}


int main()
{
	int i;
	double B[N];      // HOST
	double radius[N];    // HOST
	double *B_d;      // DEVICE
	double *radius_d; // DEVICE
	double deg = 0.0;
	FILE *outputfile;

	outputfile = fopen("output_cosine.txt", "w"); 
	if (outputfile == NULL) {
		printf("cannot open file! \n");
		exit(1);
	}

	for (int i = 0; i <= N; i+=1) {
		radius[i] = DEG_TO_RAD(deg);
		deg += 0.1;
	}

        dim3 blocks(1,1,1);
        dim3 threads(1,1,1);

	hipMalloc( (void**) &B_d, N*sizeof(double));
	hipMalloc( (void**) &radius_d, N*sizeof(double));
	
	hipMemcpy(B_d, B, N*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy(radius_d, radius, N*sizeof(double), hipMemcpyHostToDevice); 
	
	normal_cosine_function<<< blocks, threads >>>(B_d, radius_d);

        hipMemcpy(B, B_d, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0;i<=N;i+=1){
		fprintf(outputfile,"%d %.16f\n",i, B[i]);
	}

	fclose(outputfile);

        hipFree(B_d);
        hipFree(radius_d);

    return 0;
}
