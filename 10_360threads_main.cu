#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define N 3600 
#define PI 3.14159265358979323846
#define DEG_TO_RAD(deg)  ((deg) / 180.0 * (PI))

__global__ void normal_cosine_function1_256(double *B_d, double *radius_d)
{
	int t = threadIdx.x;
	int T = blockDim.x;
	for (int i = t; i <= N; i += T)
		B_d[i] = cos(radius_d[i]);
}


int main()
{
	int i;
	double B[N];      // HOST
	double radius[N];    // HOST
	double *B_d;      // DEVICE
	double *radius_d; // DEVICE
	double deg = 0.0;
	FILE *outputfile;

	outputfile = fopen("./outputs/10_360_cos.txt", "w"); 
	if (outputfile == NULL) {
		printf("cannot open file! \n");
		exit(1);
	}

	for (int i = 0; i <= N; i+=1) {
		radius[i] = DEG_TO_RAD(deg);
		deg += 0.1;
	}

        dim3 blocks(10,1,1);
        dim3 threads(360,1,1);

	hipMalloc( (void**) &B_d, N*sizeof(double));
	hipMalloc( (void**) &radius_d, N*sizeof(double));
	
	hipMemcpy(B_d, B, N*sizeof(double), hipMemcpyHostToDevice); 
	hipMemcpy(radius_d, radius, N*sizeof(double), hipMemcpyHostToDevice); 
	
	normal_cosine_function1_256<<< blocks, threads >>>(B_d, radius_d);

        hipMemcpy(B, B_d, N*sizeof(double), hipMemcpyDeviceToHost);
	
	for(i=0;i<=N;i+=1){
		fprintf(outputfile,"%d %.16f\n",i, B[i]);
	}

	fclose(outputfile);

        hipFree(B_d);
        hipFree(radius_d);

    return 0;
}
